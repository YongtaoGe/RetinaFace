#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------

#include <stdio.h>
#include <iostream>
#include <vector>
#include "gpu_nms.hpp"

#define CUDA_CHECK(condition)                                    \
    /* Code block avoids redefinition of hipError_t error */    \
    do {                                                         \
        hipError_t error = condition;                           \
        if (error != hipSuccess) {                              \
            std::cout << hipGetErrorString(error) << std::endl; \
        }                                                        \
    } while (0)

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))
#define MULTIPLIER 16
#define LONGLONG_SIZE 64

int const threadsPerBlock =
    sizeof(unsigned long long) * 8 *
    MULTIPLIER;  // number of bits for a long long variable

__device__ inline float devIoU(float const* const a, float const* const b) {
    float left = max(a[0], b[0]), right = min(a[2], b[2]);
    float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
    float width = max(right - left + 1, 0.f),
          height = max(bottom - top + 1, 0.f);
    float interS = width * height;
    float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
    float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
    return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float* dev_boxes,
                           unsigned long long* dev_mask) {
    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    // if (row_start > col_start) return;

    const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
    const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

    __shared__ float block_boxes[threadsPerBlock * 5];
    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x * 5 + 0] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
        block_boxes[threadIdx.x * 5 + 1] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
        block_boxes[threadIdx.x * 5 + 2] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
        block_boxes[threadIdx.x * 5 + 3] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
        block_boxes[threadIdx.x * 5 + 4] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
    }
    __syncthreads();

    unsigned long long ts[MULTIPLIER];

    if (threadIdx.x < row_size) {
#pragma unroll
        for (int i = 0; i < MULTIPLIER; ++i) {
            ts[i] = 0;
        }
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const float* cur_box = dev_boxes + cur_box_idx * 5;
        int i = 0;
        int start = 0;
        if (row_start == col_start) {
            start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
                ts[i / LONGLONG_SIZE] |= 1ULL << (i % LONGLONG_SIZE);
            }
        }
        const int col_blocks = DIVUP(n_boxes, threadsPerBlock);

#pragma unroll
        for (int i = 0; i < MULTIPLIER; ++i) {
            dev_mask[(cur_box_idx * col_blocks + col_start) * MULTIPLIER + i] =
                ts[i];
        }
    }
}

void _set_device(int device_id) {
    int current_device;
    CUDA_CHECK(hipGetDevice(&current_device));
    if (current_device == device_id) {
        return;
    }
    // The call to hipSetDevice must come before any calls to Get, which
    // may perform initialization using the GPU.
    CUDA_CHECK(hipSetDevice(device_id));
}

const size_t MEMORY_SIZE = 500000000;
size_t nms_Malloc() {
    float* boxes_dev = NULL;
    CUDA_CHECK(hipMalloc(&boxes_dev, MEMORY_SIZE));
    return size_t(boxes_dev);
}

void _nms(int* keep_out, int* num_out, const float* boxes_host, int boxes_num,
          int boxes_dim, float nms_overlap_thresh, int device_id, size_t base) {
    _set_device(device_id);

    float* boxes_dev = NULL;
    unsigned long long* mask_dev = NULL;

    const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

    if (base > 0) {
        size_t require_mem =
            boxes_num * boxes_dim * sizeof(float) +
            boxes_num * col_blocks * sizeof(unsigned long long) * MULTIPLIER;
        if (require_mem >= MEMORY_SIZE) {
            std::cout << "require_mem: " << require_mem << std::endl;
        }
        boxes_dev = (float*)(base);
        mask_dev =
            (unsigned long long*)(base +
                                  512 * ((unsigned long long)(boxes_num *
                                                              boxes_dim *
                                                              sizeof(float) /
                                                              512) +
                                         1));
    } else {
        CUDA_CHECK(
            hipMalloc(&boxes_dev, boxes_num * boxes_dim * sizeof(float)));
        CUDA_CHECK(hipMalloc(&mask_dev, MULTIPLIER * boxes_num * col_blocks *
                                             sizeof(unsigned long long)));
    }
    CUDA_CHECK(hipMemcpy(boxes_dev, boxes_host,
                          boxes_num * boxes_dim * sizeof(float),
                          hipMemcpyHostToDevice));

    dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
                DIVUP(boxes_num, threadsPerBlock));
    dim3 threads(threadsPerBlock);
    nms_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes_dev,
                                    mask_dev);

    std::vector<unsigned long long> mask_host(boxes_num * col_blocks *
                                              MULTIPLIER);
    CUDA_CHECK(hipMemcpy(
        &mask_host[0], mask_dev,
        sizeof(unsigned long long) * boxes_num * col_blocks * MULTIPLIER,
        hipMemcpyDeviceToHost));

    std::vector<unsigned long long> remv(col_blocks * MULTIPLIER);
    memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks * MULTIPLIER);

    int num_to_keep = 0;
    for (int i = 0; i < boxes_num; i++) {
        int nblock = i / threadsPerBlock;
        int inblock = i % threadsPerBlock;
        int offset = inblock / LONGLONG_SIZE;
        int bit_pos = inblock % LONGLONG_SIZE;

        if (!(remv[nblock * MULTIPLIER + offset] & (1ULL << bit_pos))) {
            keep_out[num_to_keep++] = i;
            unsigned long long* p = &mask_host[0] + i * col_blocks * MULTIPLIER;
            for (int j = nblock * MULTIPLIER + offset;
                 j < col_blocks * MULTIPLIER; j++) {
                remv[j] |= p[j];
            }
        }
    }
    *num_out = num_to_keep;

    if (!base) {
        CUDA_CHECK(hipFree(boxes_dev));
        CUDA_CHECK(hipFree(mask_dev));
    }
}
